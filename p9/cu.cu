#include <time.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>
#define MAX(a,b) (((a)>(b))?(a):(b))


__global__ void sumArrayKernel(int *arr, int *result, int arrLen) {
    int id = blockIdx.x * blockDim.x + threadIdx.x, sum = 0;

    if (id < arrLen) sum = arr[id];

    __shared__ int sumArr[256];
    sumArr[threadIdx.x] = sum;
    __syncthreads();

    for (int i = blockDim.x / 2; i > 0; i>>=1) {
        int idInBlock = threadIdx.x;
        if (idInBlock < i) sumArr[idInBlock] += sumArr[idInBlock + i];
        __syncthreads();
    }

    if (threadIdx.x == 0) atomicAdd(result, sumArr[0]);
}

/**
 * @brief Set thread start index and how many numbers to sum
 *
 * This function computes the start and amount for thread to calculate sum in array
 *
 * @param threadNum Index of thread
 * @param numThreads Total number of executing threads
 * @param arrLen Length of array
 * @param range Pointer to start of range array
 */
void getThreadRange(int threadNum, int numThreads, int arrLen, int* range) {
    int sliceLen = arrLen / numThreads;
    range[0] = threadNum * sliceLen;
    range[1] = threadNum == numThreads - 1 ? arrLen - range[0] : (arrLen < range[0] ? 1 : sliceLen);
}


/**
 * @brief Sum all numbers inside an array
 *
 * This function computes the sum of every integer inside an array
 *
 * @param arrLen Length of array
 * @param arr Pointer to start of array
 *
 * @return Sum of integers in array
 */
int sumArray(unsigned int arrLen, int* arr) {
    if (arrLen == 0) {
        free(arr);
        abort();
    }

    int counter = 0;
    for (int i = 0; i < arrLen; i++) {
        counter += arr[i];
    }
    return counter;
}


/**
 * @brief Get random array from time(NULL)
 *
 * This function computes a N sized array with random integers from min to max (exclusive)
 *
 * @param n Size of array
 * @param arr Array to put integers into
 * @param min Minimum integer (inclusive)
 * @param max Max integer (exclusive)
 */
void randomArray(unsigned int n, int *arr, int min, int max) {
    srand(time(NULL));

    if (n == 0 || min > max) {
        free(arr);
        abort();
    }

    #pragma omp target map(to: arr[0:n])
    {
        #pragma omp parallel for 
        for (int i=0; i < n; i++) {
            arr[i] = rand() % (max - min) + min;
        }
    }
}


/**
 * @brief Adds all numbers inside an array with multiple threads
 *
 * This function computes the sum of every integer inside an array using multiple threads
 *
 * @param arrLen Length of array
 * @param arr Pointer to start of array
 *
 * @return Sum of integers in array
 */
int sumArrayMultithread(unsigned int arrLen, int *arr, unsigned int numThreads) {

    if (numThreads == 0 || arrLen == 0) {
        free(arr);
        abort();
    }

    int collector = 0;

    #pragma omp parallel num_threads(numThreads) reduction(+:collector)
    {
        int threadNum = omp_get_thread_num();

        int threadRangeArr[2];
        getThreadRange(threadNum, numThreads, arrLen, threadRangeArr);
        #pragma omp atomic
        collector += sumArray(threadRangeArr[1], &arr[threadRangeArr[0]]);
    }

    return collector;
}


/**
 * @brief Print an array of integers
 *
 * This function prints an array of integers to stdin
 *
 * @param arrLen Length of array
 * @param arr Pointer to start of array
 */
void printArrayString(unsigned int arrLen, int* arr) {
    printf("[ ");
    for (int i = 0; i < arrLen; i++) {
        printf("%d ", arr[i]);
    }
    printf("]\n");
}


int main(int argc, char* argv[]) {
    unsigned int arrLen = (unsigned int)atoi(argv[1]);

    int *arr = (int *)malloc(arrLen * sizeof(int));
    randomArray(arrLen, arr, -100, 100);
    printArrayString(arrLen, arr);

    int *dArr, *dRes;
    int collector= 0;

    hipMalloc((void **)&dArr, arrLen * sizeof(int));
    hipMalloc((void **)&dRes, arrLen * sizeof(int));

    hipMemcpy(dRes, &collector, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dArr, arr, arrLen * sizeof(int), hipMemcpyHostToDevice);


    int numThreads = 256;
    int numBlocks = (arrLen + numThreads - 1) / numThreads;
    double s = omp_get_wtime();
    sumArrayKernel<<<numBlocks, numThreads>>>(dArr, dRes, arrLen);
    double ss = omp_get_wtime();
    hipDeviceSynchronize();

    hipMemcpy(&collector, dRes, sizeof(int), hipMemcpyDeviceToHost);

    hipFree(dArr);
    hipFree(dRes);

    printf("\nN threads used: %d\nSum: %d\nElapsed time: %.8f\n\n", numThreads, collector, ss - s);

    s = omp_get_wtime();
    collector = sumArray(arrLen, arr);
    ss = omp_get_wtime();

    printf("N threads used: 1\nSum: %d\nElapsed time: %.8f\n", collector, omp_get_wtime() - s);
    free(arr);
    return 0;
}

